#include "hip/hip_runtime.h"
#include "../AnalysisLib/CFD.hpp"
#include <hip/hip_runtime.h>
#include <memory>
#include "../LibCo/print.hpp"
#include "../LibCo/Timer.hpp"
#include "../LibCo/randomCo.hpp"

/*
So, overall it works. However the process is mainly sped up if the resources are correcly managed.
*/

__global__
void batch_cfd_kernel(const double* traces, double* cfds, int shift, double fraction, int n_samples, int output_samples) {
    int trace_idx = blockIdx.x;
    int thread_idx = threadIdx.x;
    int stride = blockDim.x;

    const double* trace = &traces[trace_idx * n_samples];
    double* cfd = &cfds[trace_idx * output_samples];

    for (int i = thread_idx; i < output_samples; i += stride) {
        int bin = i + 2 * shift;
        cfd[i] = fraction * trace[bin] - trace[bin - shift];
    }
}


class BatchCFD {
public:
  BatchCFD() noexcept = default;

  BatchCFD(const std::vector<std::vector<double>>& traceVec, int shift, double fraction)
      : n_traces(traceVec.size()), shift(shift), fraction(fraction)
  {
    n_samples = traceVec[0].size();  // Assume all traces are same length
    output_samples = n_samples - 2 * shift;

    size_t trace_bytes = sizeof(double) * n_traces * n_samples;
    size_t cfd_bytes = sizeof(double) * n_traces * output_samples;

    std::vector<double> trace_flat(n_traces * n_samples);
    for (size_t i = 0; i < n_traces; ++i)
        std::copy(traceVec[i].begin(), traceVec[i].end(), trace_flat.begin() + i * n_samples);

    // Allocate and copy to GPU
    hipMalloc(&d_traces, trace_bytes);
    hipMalloc(&d_cfds, cfd_bytes);
    hipMemcpy(d_traces, trace_flat.data(), trace_bytes, hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    batch_cfd_kernel<<<n_traces, threadsPerBlock>>>(d_traces, d_cfds, shift, fraction, n_samples, output_samples);
    hipDeviceSynchronize();

    // Copy results back
    cfds.resize(n_traces * output_samples);
    hipMemcpy(cfds.data(), d_cfds, cfd_bytes, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_traces);
    hipFree(d_cfds);
  }

  std::vector<std::vector<double>> getResults() const 
  {
    std::vector<std::vector<double>> result(n_traces, std::vector<double>(output_samples));
    for (size_t i = 0; i < n_traces; ++i) 
      std::copy(cfds.begin() + i * output_samples, cfds.begin() + (i + 1) * output_samples, result[i].begin());
    return result;
  }

  std::vector<CFD*> cfds;

private:

  int n_traces, n_samples, shift, output_samples;
  double fraction;

  double* d_traces = nullptr;
  double* d_cfds = nullptr;
  std::vector<double> cfds;
};


int main() {
  int n_traces = 10000;
  int n_samples = 512;
  int shift = 2;
  double fraction = 0.5;

  std::vector<CFD*> traces(n_traces);
  BatchCFD cfds;
  for (int i = 0; i < n_traces; ++i)
  {
    traces[i] = new CFD();
    traces[i]->cfd.resize(n_samples);
    for (int j = 0; j < n_samples; ++j) traces[i]->cfd[j] = std::sin(j / 10.0 + i * 0.001);
    cfds.cfds.push_back(traces[i]);
  }
  
  {
    BatchCFD cfds(traces, shift, fraction);
    Timer timer;
    auto results = cfds.getResults();
    std::cout << "Time: " << timer() << " seconds\n";
  }

  for (auto const & trace : traces) cfd_o.push_back(trace);

  Timer timer;
  for (auto & cfd : cfd_o) cfd.calculate(shift, fraction);
  std::cout << "Time: " << timer() << " seconds\n";
}

//nvcc -Xptxas -O3,-v -std=c++17 -o cfd_gpu test.cu 